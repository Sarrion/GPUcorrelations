
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include <stdlib.h>
#include "GPUcorrelationsFunctions.h"

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



int main()
{
	/*_______reading_data_____________*/
	matrx M = readCSV("C:\\Users\\AVSM2\\Documents\\datos.txt");
	displayMatrix(readCSV("C:\\Users\\AVSM2\\Documents\\datos.txt"));

	///*_______correlation_matrix_______*/
	//matrx cors = corr(M);
	//printf("\n");
	//printf("Correlation Matrix:\n\n");
	//displayMatrix(cors);

	/*_______save_correlation_matrix__*/
	//writeCSV(cors, "C:\\Users\\AVSM2\\Documents\\datos_correlation_matrix_GPU.txt");

	double** Result;
	Result = (double**)malloc(sizeof(double*) * M.ncol);
	for (int row = 0; row < M.nrow; row++) {
		Result[row] = (double*)malloc(sizeof(double) * M.ncol);
	}


    // Correlations in parallel.
    //hipError_t cudaStatus = GPUcorr(c, a, b, arraySize);
	hipError_t cudaStatus = GPUcorr(M, Result);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GPUcorr failed!");
        return 1;
    }
	
	matrx matrx_Result = { M.ncol, M.ncol, Result };
	printf("lalalala:\n\n");
	displayMatrix(matrx_Result);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


